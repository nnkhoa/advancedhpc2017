#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2017, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
            printf("labwork 1 CPU-OMP ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {		// let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3)); 
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
	#pragma omp parallel for schedule(dynamic)
	    for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    int deviceCount;
    struct hipDeviceProp_t gpuProp;

    hipGetDeviceCount(&deviceCount);
    
    printf("number of GPU: %d\n", deviceCount);

    for(int i = 0; i < deviceCount; i++){
        hipGetDeviceProperties(&gpuProp, i);
        printf("Name: %s\n", gpuProp.name);
        printf("Max Threads per Block: %d\n", gpuProp.maxThreadsPerBlock);
        printf("Total Core: %d\n", getSPcores(gpuProp));
        printf("Clock Rate: %d\n", gpuProp.clockRate);
        printf("Multi Processor Count: %d\n", gpuProp.multiProcessorCount);
        printf("Warp Size: %d\n", gpuProp.warpSize);
        printf("Memory Clock Rate: %d\n", gpuProp.memoryClockRate);
        printf("Memory Bus Width: %d\n", gpuProp.memoryBusWidth);
        printf("Memory Bandwidth: %d\n", gpuProp.memoryClockRate*gpuProp.memoryBusWidth);
    }

}

__global__ void grayscaleConvert(char* input, char* output, int imagePixelCount){
        for (int i = 0; i < imagePixelCount; i++) {
            output[i * 3] = (char) (((int) input[i * 3] + (int) input[i * 3 + 1] +
                                          (int) input[i * 3 + 2]) / 3);
            output[i * 3 + 1] = output[i * 3];
            output[i * 3 + 2] = output[i * 3];
        }
}

void Labwork::labwork3_GPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3)); 

    char *blockSizeEnv = getenv("CUDA_BLOCK_SIZE");

    if(!blockSizeEnv){
        printf("No Environment Variable specified\n");
        printf("Please use > CUDA_BLOCK_SIZE=block_size ./labwork ...\n");
        return;
    }

    int blockSize = atoi(blockSizeEnv);
    long numBlock = pixelCount/blockSize;

    char *cuInput, *cuOutput;
    hipMalloc(&cuInput, pixelCount*3*sizeof(char));
    hipMalloc(&cuOutput, pixelCount*3*sizeof(char));
    
    hipMemcpy(cuInput, inputImage->buffer, pixelCount*3*sizeof(char), hipMemcpyHostToDevice);
    
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
    	grayscaleConvert<<<numBlock, blockSize>>>(cuInput, cuOutput, pixelCount);
    }
    hipMemcpy(outputImage, cuOutput, pixelCount*3*sizeof(char), hipMemcpyDeviceToHost);
    
    hipFree(cuOutput);
    hipFree(cuInput);
}

void Labwork::labwork4_GPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3)); 

    if(!blockSizeEnv){
        printf("No Environment Variable specified\n");
        printf("Please use > CUDA_BLOCK_SIZE=block_size ./labwork ...\n");
        return;
    }

    dim3 gridSize = dime3(8,8);
    dim3 blockSize = dime3(32,32);

    char *cuInput, *cuOutput;
    hipMalloc(&cuInput, pixelCount*3*sizeof(char));
    hipMalloc(&cuOutput, pixelCount*3*sizeof(char));
    
    hipMemcpy(cuInput, inputImage->buffer, pixelCount*3*sizeof(char), hipMemcpyHostToDevice);
    
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        grayscaleConvert<<<gridSize, blockSize>>>(cuInput, cuOutput, pixelCount);
    }
    hipMemcpy(outputImage, cuOutput, pixelCount*3*sizeof(char), hipMemcpyDeviceToHost);
    
    hipFree(cuOutput);
    hipFree(cuInput);   
}

void Labwork::labwork5_GPU() {
    
}

void Labwork::labwork6_GPU() {

}

void Labwork::labwork7_GPU() {

}

void Labwork::labwork8_GPU() {

}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU() {

}
